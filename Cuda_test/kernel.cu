﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

// cuda kernel
__global__ void add(int* a, int* b, int* c) {

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    c[i] = a[i] + b[i];


}


__managed__ int vector_a[256], vector_b[256], vector_c[256];

int main() {

    for (int i = 0; i < 256; i++) {

        vector_a[i] = i;
        vector_b[i] = 256 - i;
    }

    add <<< 1, 256 >>> (vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    int result_sum = 0;

    for (int i = 0; i < 256; i++) {
        result_sum += vector_c[i];
    }

    printf("The result of these vectors \n");
    printf("Result: %d ", result_sum);
    return 0;
}